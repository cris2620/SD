#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

const int N = 1*1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = ceil(n/2560);

// vector sum C = A + B
// Each thread performs one pair wise addition
__global__
void vecAddKernel(float* A, float* B, float* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n){
    int size = n* sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &B_d, size);
    ccudaMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_C, size);
    vecAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    // Free device memory for A, B, C
    hipFree(d_Ad); hipFree(d_B); hipFree(d_C);
}

int main( void ) {
    cout << "Suma de vectores" << endl;
    float *h_A, *h_B, *h_C;  

    h_A = new float[N];
    h_B = new float[N];
    h_C = new float[N];

    for (int i=0; i<N; i++) {
        h_A[i] = i;
        h_B[i] = i+1;
    }

    vecAdd(h_A, h_B, h_C, N);

    for (int i=0; i<N; i++) {
        cout << h_C[i] << endl;
    }

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}